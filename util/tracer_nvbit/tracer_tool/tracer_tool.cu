#include "hip/hip_runtime.h"
/* Author1: Mahmoud Khairy, abdallm@purdue.com - 2019 */
/* Author2: Jason Shen, shen203@purdue.edu - 2019 */

#include <assert.h>
#include <inttypes.h>
#include <stdint.h>
#include <stdio.h>
#include <sys/stat.h>
#include <unistd.h>

#include <algorithm>
#include <bitset>
#include <iostream>
#include <iterator>
#include <map>
#include <regex>
#include <sstream>
#include <string>
#include <unordered_map>
#include <unordered_set>
#include <vector>
/* every tool needs to include this once */
#include "nvbit_tool.h"

/* nvbit interface file */
#include "nvbit.h"

/* for channel */
#include "utils/channel.hpp"

/* contains definition of the inst_trace_t structure */
#include "common.h"

#define TRACER_VERSION "5"

/* Channel used to communicate from GPU to CPU receiving thread */
#define CHANNEL_SIZE (1l << 20)
static __managed__ ChannelDev channel_dev;
static ChannelHost channel_host;

/* receiving thread and its control variables */
pthread_t recv_thread;
volatile bool recv_thread_started = false;
volatile bool recv_thread_receiving = false;

/* skip flag used to avoid re-entry on the nvbit_callback when issuing
 * flush_channel kernel call */
bool skip_flag = false;

/* global control variables for this tool */
uint32_t instr_begin_interval = 0;
uint32_t instr_end_interval = UINT32_MAX;
int verbose = 0;
int enable_compress = 1;
int print_core_id = 0;
int exclude_pred_off = 1;
int active_from_start = 1;
int lineinfo = 0;
/* used to select region of interest when active from start is 0 */
bool active_region = true;

/* Should we terminate the program once we are done tracing? */
int terminate_after_limit_number_of_kernels_reached = 0;
int user_defined_folders = 0;

/* Use xz to compress the *.trace file */
int xz_compress_trace = 0;

/* opcode to id map and reverse map  */
std::map<std::string, int> opcode_to_id_map;
std::map<int, std::string> id_to_opcode_map;

std::string user_folder = getcwd(NULL, 0);
std::string cwd = getcwd(NULL, 0);
std::string traces_location = cwd + "/traces/";
std::string kernelslist_location = cwd + "/traces/kernelslist";
std::string stats_location = cwd + "/traces/stats.csv";

std::unordered_map<hipCtx_t, std::string> ctx_kernelslist;
std::unordered_map<hipCtx_t, std::string> ctx_stats_location;
std::unordered_map<hipCtx_t, int> ctx_kernelid;
std::unordered_map<hipCtx_t, FILE *> ctx_resultsFile;

std::string kernel_ranges = "";

struct KernelRange {
  uint64_t start;
  uint64_t end; // UINT64_MAX means open-ended
  std::vector<std::regex>
      kernel_name_regexes; // Vector of regexes for multiple patterns
};
std::vector<KernelRange> g_kernel_ranges;
uint64_t g_max_kernel_id = 0;
void parse_kernel_ranges_from_env() {
  g_kernel_ranges.clear();
  g_max_kernel_id = 0;

  const char *env_var = std::getenv("DYNAMIC_KERNEL_RANGE");
  if (!env_var || std::string(env_var).empty()) {
    g_kernel_ranges.push_back({0, 0, {std::regex(".*")}}); // 0 end = trace all
    return;
  }
  std::string input(env_var);
  std::istringstream stream(input);
  std::string token;

  while (stream >> token) {
    if (token.empty())
      continue;

    uint64_t start = 0, end = 0;
    std::vector<std::regex> regexes;

    size_t at_pos = token.find('@');
    std::string range_part, regex_part;

    if (at_pos != std::string::npos) {
      range_part = token.substr(0, at_pos);
      regex_part = token.substr(at_pos + 1);
    } else {
      range_part = token;
    }

    // Parse the range
    if (!range_part.empty()) {
      size_t dash_pos = range_part.find('-');
      if (dash_pos != std::string::npos) {
        std::string start_str = range_part.substr(0, dash_pos);
        std::string end_str = range_part.substr(dash_pos + 1);

        start = std::stoull(start_str);
        if (!end_str.empty()) {
          end = std::stoull(end_str);
        } else {
          end = 0; // open-ended
        }
      } else {
        start = std::stoull(range_part);
        end = start;
      }
    } else {
      // No range → match all IDs
      start = 0;
      end = 0;
    }

    // Parse the regexes
    if (!regex_part.empty()) {
      std::istringstream regex_stream(regex_part);
      std::string regex_token;
      while (std::getline(regex_stream, regex_token, ',')) {
        try {
          regexes.emplace_back(regex_token);
        } catch (const std::regex_error &e) {
          std::cerr << "Invalid regex: " << regex_token << std::endl;
        }
      }
    } else {
      regexes.emplace_back(".*"); // match all kernel names
    }

    g_kernel_ranges.push_back({start, end, regexes});
    if (end > g_max_kernel_id) {
      g_max_kernel_id = end;
    }
  }
}

bool should_trace_kernel(uint64_t kernel_id, const std::string &kernel_name) {
  for (const auto &range : g_kernel_ranges) {
    // Check range for kernel ID
    if (range.end == 0) {
      if (kernel_id >= range.start) {
        // Match any of the regexes for this range
        for (const auto &regex : range.kernel_name_regexes) {
          if (std::regex_match(kernel_name, regex)) {
            return true;
          }
        }
      }
    } else if (kernel_id >= range.start && kernel_id <= range.end) {
      // Match any of the regexes for this range
      for (const auto &regex : range.kernel_name_regexes) {
        if (std::regex_match(kernel_name, regex)) {
          return true;
        }
      }
    }
  }
  return false;
}

enum address_format { list_all = 0, base_stride = 1, base_delta = 2 };

/* File pointers for the kernels, and stats files */
static FILE *kernelsFile = NULL;
static FILE *statsFile = NULL;
static bool first_call = true;

unsigned old_total_insts = 0;
unsigned old_total_reported_insts = 0;

void nvbit_at_init() {
  setenv("CUDA_MANAGED_FORCE_DEVICE_ALLOC", "1", 1);
  GET_VAR_INT(
      instr_begin_interval, "INSTR_BEGIN", 0,
      "Beginning of the instruction interval where to apply instrumentation");
  GET_VAR_INT(instr_end_interval, "INSTR_END", UINT32_MAX,
              "End of the instruction interval where to apply instrumentation");
  GET_VAR_INT(exclude_pred_off, "EXCLUDE_PRED_OFF", 1,
              "Exclude predicated off instruction from count");
  GET_VAR_INT(lineinfo, "TRACE_LINEINFO", 0,
              "Include source code line info at the start of each traced line. "
              "The target binary must be compiled with -lineinfo or "
              "--generate-line-info");
  GET_VAR_STR(
      kernel_ranges, "DYNAMIC_KERNEL_RANGE",
      "Specify kernel IDs or ranges to trace. Format:\n"
      "  - Single ID:       \"2\" traces only kernel 2.\n"
      "  - Range:           \"5-8\" traces kernels 5 through 8 (inclusive).\n"
      "  - Open-ended:      \"10-\" traces from kernel 10 onward.\n"
      "  - Multiple ranges: \"2 5-8 10-\" (space-separated).\n"
      "  - With regex:      \"5-8@kernel_a.*,kernel_b.*\" traces kernels 5–8 "
      "with matching names.\n"
      "If unset or empty, all kernels will be traced from the beginning.");
  GET_VAR_INT(
      active_from_start, "ACTIVE_FROM_START", 1,
      "Start instruction tracing from start or wait for hipProfilerStart "
      "and hipProfilerStop. If set to 0, DYNAMIC_KERNEL_RANGE options have no "
      "effect");
  GET_VAR_INT(verbose, "TOOL_VERBOSE", 0, "Enable verbosity inside the tool");
  GET_VAR_INT(enable_compress, "TOOL_COMPRESS", 1, "Enable traces compression");
  GET_VAR_INT(print_core_id, "TOOL_TRACE_CORE", 0,
              "write the core id in the traces");
  GET_VAR_INT(
      terminate_after_limit_number_of_kernels_reached, "TERMINATE_UPON_LIMIT",
      0, "Stop the process once the current kernel > DYNAMIC_KERNEL_LIMIT_END");
  GET_VAR_INT(user_defined_folders, "USER_DEFINED_FOLDERS", 0,
              "Uses the user defined "
              "folder TRACES_FOLDER path environment");
  GET_VAR_INT(xz_compress_trace, "TRACE_FILE_COMPRESS", 1,
              "Create xz-compressed trace"
              "file");
  std::string pad(100, '-');
  printf("%s\n", pad.c_str());

  active_region = false;
  char *usr_defined_folder = std::getenv("TRACES_FOLDER");
  if (usr_defined_folder != NULL)
    user_folder = usr_defined_folder;
  parse_kernel_ranges_from_env();
}

/* Set used to avoid re-instrumenting the same functions multiple times */
std::unordered_set<hipFunction_t> already_instrumented;

/* instrument each memory instruction adding a call to the above instrumentation
 * function */
void instrument_function_if_needed(hipCtx_t ctx, hipFunction_t func) {
  std::vector<hipFunction_t> related_functions =
      nvbit_get_related_functions(ctx, func);

  /* add kernel itself to the related function vector */
  related_functions.push_back(func);

  /* iterate on function */
  for (auto f : related_functions) {
    /* "recording" function was instrumented, if set insertion failed
     * we have already encountered this function */
    if (!already_instrumented.insert(f).second) {
      continue;
    }

    const std::vector<Instr *> &instrs = nvbit_get_instrs(ctx, f);
    if (verbose >= 1) {
      printf("Inspecting function %s at address 0x%lx\n",
             nvbit_get_func_name(ctx, f), nvbit_get_func_addr(ctx, f));
    }

    uint32_t cnt = 0;
    /* iterate on all the static instructions in the function */
    for (auto instr : instrs) {
      uint32_t line_num = 0;
      // Temporary workaround for a bug in NVBit 1.7.4, which does not correctly
      // handle `call.rel`. Instrumenting this instruction leads to illegal
      // memory access. Refer to:
      // https://github.com/NVlabs/NVBit/issues/142#issue-2911561744
      if (!strcmp(instr->getOpcode(), "CALL.REL.NOINC")) {
        printf("Warning: Ignoring CALL.REL.NOINC (NVBit 1.7.4 bug)\n");
        continue;
      }

      if (cnt < instr_begin_interval || cnt >= instr_end_interval) {
        cnt++;
        continue;
      }

      if (verbose >= 2) {
        instr->printDecoded();
      }

      if (lineinfo) {
        char *file_name, *dir_name;
        nvbit_get_line_info(ctx, func, instr->getOffset(), &file_name,
                            &dir_name, &line_num);
      }

      if (opcode_to_id_map.find(instr->getOpcode()) == opcode_to_id_map.end()) {
        int opcode_id = opcode_to_id_map.size();
        opcode_to_id_map[instr->getOpcode()] = opcode_id;
        id_to_opcode_map[opcode_id] = instr->getOpcode();
      }

      int opcode_id = opcode_to_id_map[instr->getOpcode()];

      /* check all operands. For now, we ignore constant, TEX, predicates and
       * unified registers. We only report vector regisers */
      int src_oprd[MAX_SRC];
      int srcNum = 0;
      int dst_oprd = -1;
      int mem_oper_idx = -1;
      int num_mref = 0;
      uint64_t imm_value = 0;

      for (int i = 0; i < instr->getNumOperands(); ++i) {
        const InstrType::operand_t *op = instr->getOperand(i);
        if (op->type == InstrType::OperandType::MREF) {
          assert(srcNum < MAX_SRC);
          src_oprd[srcNum] = instr->getOperand(i)->u.mref.ra_num;
          srcNum++;
          mem_oper_idx++;
          num_mref++;
          // if(mem_oper_idx == 0){
          //   mem_oper_idx = 1; // loop control
          // }
        } else if (op->type == InstrType::OperandType::REG) {
          if (i == 0) {
            // find dst reg
            dst_oprd = instr->getOperand(0)->u.reg.num;
          } else {
            // find src regs
            assert(srcNum < MAX_SRC);
            src_oprd[srcNum] = instr->getOperand(i)->u.reg.num;
            srcNum++;
          }
        }
        // Add immediate value for DEPBAR instruction
        else if (op->type == InstrType::OperandType::IMM_UINT64) {
          imm_value = instr->getOperand(i)->u.imm_uint64.value;
        }
      }

      do {
        /* insert call to the instrumentation function with its
         * arguments */
        nvbit_insert_call(instr, "instrument_inst", IPOINT_BEFORE);

        /* pass predicate value */
        nvbit_add_call_arg_guard_pred_val(instr);

        /* send opcode and pc */
        nvbit_add_call_arg_const_val32(instr, opcode_id);
        nvbit_add_call_arg_const_val32(instr, (int)instr->getOffset());

        /* mem addresses info */
        if (mem_oper_idx >= 0) {
          nvbit_add_call_arg_const_val32(instr, 1);
          assert(num_mref <= 2);
          if (num_mref == 2) { // LDGSTS
            nvbit_add_call_arg_mref_addr64(instr, 1 - mem_oper_idx);
          } else {
            nvbit_add_call_arg_mref_addr64(instr, mem_oper_idx);
          }
          nvbit_add_call_arg_const_val32(instr, (int)instr->getSize());
        } else {
          nvbit_add_call_arg_const_val32(instr, 0);
          nvbit_add_call_arg_const_val64(instr, static_cast<uint64_t>(-1));
          nvbit_add_call_arg_const_val32(instr, static_cast<uint32_t>(-1));
        }

        /* reg info */
        nvbit_add_call_arg_const_val32(instr, dst_oprd);
        for (int i = 0; i < srcNum; i++) {
          nvbit_add_call_arg_const_val32(instr, src_oprd[i]);
        }
        for (int i = srcNum; i < MAX_SRC; i++) {
          nvbit_add_call_arg_const_val32(instr, static_cast<uint32_t>(-1));
        }
        nvbit_add_call_arg_const_val32(instr, srcNum);

        /* immediate info */
        nvbit_add_call_arg_const_val64(instr, imm_value);

        /* add pointer to channel_dev and other counters*/
        nvbit_add_call_arg_const_val64(instr, (uint64_t)&channel_dev);
        nvbit_add_call_arg_const_val64(instr,
                                       (uint64_t)&total_dynamic_instr_counter);
        nvbit_add_call_arg_const_val64(
            instr, (uint64_t)&reported_dynamic_instr_counter);
        nvbit_add_call_arg_const_val64(instr, (uint64_t)&stop_report);
        /* Add Source code line number for current instr */
        nvbit_add_call_arg_const_val32(instr, (int)line_num);

        mem_oper_idx--;
      } while (mem_oper_idx >= 0);

      cnt++;
    }
  }
}

__global__ void flush_channel() {
  /* push memory access with negative cta id to communicate the kernel is
   * completed */
  inst_trace_t ma;
  ma.cta_id_x = -1;
  channel_dev.push(&ma, sizeof(inst_trace_t));

  /* flush channel */
  channel_dev.flush();
}

static void enter_kernel_launch(hipCtx_t ctx, hipFunction_t func,
                                nvbit_api_cuda_t cbid, void *params,
                                bool stream_capture = false,
                                bool build_graph = false) {
  // no need to sync during stream capture or manual graph build, since no
  // kernel is actually launched.
  if (!stream_capture && !build_graph) {
    /* Make sure GPU is idle */
    hipDeviceSynchronize();
    assert(hipGetLastError() == hipSuccess);
  }

  // Mark if the kernel should be traced
  std::string func_name = std::string(nvbit_get_func_name(ctx, func, true));
  if (active_from_start && should_trace_kernel(ctx_kernelid[ctx], func_name))
    active_region = true;

  // Terminate tracing if the limit number of kernels is reached
  if (terminate_after_limit_number_of_kernels_reached && g_max_kernel_id != 0 &&
      ctx_kernelid[ctx] > g_max_kernel_id) {
    exit(0);
  }

  // Get launch config for this kernel
  unsigned int gridDimX, gridDimY, gridDimZ;
  unsigned int blockDimX, blockDimY, blockDimZ;
  unsigned int sharedMemBytes;
  hipStream_t hStream;
  if (cbid == API_CUDA_cuLaunchKernelEx_ptsz ||
      cbid == API_CUDA_cuLaunchKernelEx) {
    cuLaunchKernelEx_params *p = (cuLaunchKernelEx_params *)params;
    gridDimX = p->config->gridDimX;
    gridDimY = p->config->gridDimY;
    gridDimZ = p->config->gridDimZ;
    blockDimX = p->config->blockDimX;
    blockDimY = p->config->blockDimY;
    blockDimZ = p->config->blockDimZ;
    sharedMemBytes = p->config->sharedMemBytes;
    hStream = p->config->hStream;
  } else {
    cuLaunchKernel_params *p = (cuLaunchKernel_params *)params;
    gridDimX = p->gridDimX;
    gridDimY = p->gridDimY;
    gridDimZ = p->gridDimZ;
    blockDimX = p->blockDimX;
    blockDimY = p->blockDimY;
    blockDimZ = p->blockDimZ;
    sharedMemBytes = p->sharedMemBytes;
    hStream = p->hStream;
  }

  // Get the number of registers and shared memory size for the kernel
  int nregs;
  CUDA_SAFECALL(hipFuncGetAttribute(&nregs, HIP_FUNC_ATTRIBUTE_NUM_REGS, func));

  int shmem_static_nbytes;
  CUDA_SAFECALL(hipFuncGetAttribute(&shmem_static_nbytes,
                                   HIP_FUNC_ATTRIBUTE_SHARED_SIZE_BYTES, func));

  int binary_version;
  CUDA_SAFECALL(hipFuncGetAttribute(&binary_version,
                                   HIP_FUNC_ATTRIBUTE_BINARY_VERSION, func));

  // Instrument the kernel if needed
  instrument_function_if_needed(ctx, func);

  // Enable or disable tracing based on the active region
  if (active_region) {
    nvbit_enable_instrumented(ctx, func, true);
    stop_report = false;
  } else {
    nvbit_enable_instrumented(ctx, func, false);
    stop_report = true;
  }

  // Create the trace file per kernel
  char buffer[2048];
  std::string trace_filename = user_folder + "/traces/";
  sprintf(buffer, "%s/kernel-%d-ctx_0x%lx.trace", trace_filename.c_str(),
          ctx_kernelid[ctx], ctx);

  if (!stop_report) {
    if (!xz_compress_trace) {
      ctx_resultsFile[ctx] = fopen(buffer, "w");
      printf("Writing results to %s\n", buffer);
    } else {
      char cmd_buffer[1039];
      sprintf(cmd_buffer, "xz -1 -T0 > %s.xz", buffer);
      ctx_resultsFile[ctx] = popen(cmd_buffer, "w");
      printf("Writing results to %s.xz\n", buffer);
    }

    // Writing header information
    fprintf(ctx_resultsFile[ctx], "-kernel name = %s\n",
            nvbit_get_func_name(ctx, func, true));
    fprintf(ctx_resultsFile[ctx], "-kernel id = %d\n", ctx_kernelid[ctx]);
    fprintf(ctx_resultsFile[ctx], "-grid dim = (%d,%d,%d)\n", gridDimX,
            gridDimY, gridDimZ);
    fprintf(ctx_resultsFile[ctx], "-block dim = (%d,%d,%d)\n", blockDimX,
            blockDimY, blockDimZ);
    fprintf(ctx_resultsFile[ctx], "-shmem = %d\n",
            shmem_static_nbytes + sharedMemBytes);
    fprintf(ctx_resultsFile[ctx], "-nregs = %d\n", nregs);
    fprintf(ctx_resultsFile[ctx], "-binary version = %d\n", binary_version);
    fprintf(ctx_resultsFile[ctx], "-cuda stream id = %lu\n", (uint64_t)hStream);
    fprintf(ctx_resultsFile[ctx], "-shmem base_addr = 0x%016lx\n",
            (uint64_t)nvbit_get_shmem_base_addr(ctx));
    fprintf(ctx_resultsFile[ctx], "-local mem base_addr = 0x%016lx\n",
            (uint64_t)nvbit_get_local_mem_base_addr(ctx));
    fprintf(ctx_resultsFile[ctx], "-nvbit version = %s\n", NVBIT_VERSION);
    fprintf(ctx_resultsFile[ctx], "-accelsim tracer version = %s\n",
            TRACER_VERSION);
    fprintf(ctx_resultsFile[ctx], "-enable lineinfo = %d\n", lineinfo);
    fprintf(ctx_resultsFile[ctx], "\n");

    fprintf(ctx_resultsFile[ctx],
            "#traces format = [line_num] PC mask dest_num [reg_dests] "
            "opcode src_num "
            "[reg_srcs] mem_width [adrrescompress?] [mem_addresses] "
            "immediate\n");
    fprintf(ctx_resultsFile[ctx], "\n");
  }

  kernelsFile = fopen(ctx_kernelslist[ctx].c_str(), "a");
  // This will be a relative path to the traces file

  sprintf(buffer, "kernel-%d-ctx_0x%lx.trace%s", ctx_kernelid[ctx], ctx,
          xz_compress_trace ? ".xz" : "");
  if (!stop_report) {
    fprintf(kernelsFile, buffer);
    fprintf(kernelsFile, "\n");
  }
  fclose(kernelsFile);

  statsFile = fopen(ctx_stats_location[ctx].c_str(), "a");
  unsigned blocks = gridDimX * gridDimY * gridDimZ;
  unsigned threads = blockDimX * blockDimY * blockDimZ;

  fprintf(statsFile, "%s, %s, %d, %d, %d, %d, %d, %d, %d, %d, ", buffer,
          nvbit_get_func_name(ctx, func, true), gridDimX, gridDimY, gridDimZ,
          blocks, blockDimX, blockDimY, blockDimZ, threads);

  fclose(statsFile);

  ctx_kernelid[ctx]++;
  recv_thread_receiving = true;
}

static void leave_kernel_launch(hipCtx_t ctx, hipFunction_t func) {
  /* make sure current kernel is completed */
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("cuda error: %s\n", hipGetErrorName(err));
  }
  assert(err == hipSuccess);

  /* make sure we prevent re-entry on the nvbit_callback when issuing
   * the flush_channel kernel */
  skip_flag = true;

  /* issue flush of channel so we are sure all the memory accesses
   * have been pushed */
  flush_channel<<<1, 1>>>();
  hipDeviceSynchronize();
  assert(hipGetLastError() == hipSuccess);

  /* unset the skip flag */
  skip_flag = false;

  /* wait here until the receiving thread has not finished with the
   * current kernel */
  while (recv_thread_receiving) {
    pthread_yield();
  }

  unsigned total_insts_per_kernel =
      total_dynamic_instr_counter - old_total_insts;
  old_total_insts = total_dynamic_instr_counter;

  unsigned reported_insts_per_kernel =
      reported_dynamic_instr_counter - old_total_reported_insts;
  old_total_reported_insts = reported_dynamic_instr_counter;

  statsFile = fopen(ctx_stats_location[ctx].c_str(), "a");
  fprintf(statsFile, "%d,%d", total_insts_per_kernel,
          reported_insts_per_kernel);
  fprintf(statsFile, "\n");
  fclose(statsFile);

  if (!stop_report) {
    if (!xz_compress_trace) {
      fclose(ctx_resultsFile[ctx]);
    } else {
      pclose(ctx_resultsFile[ctx]);
    }
  }

  std::string func_name = std::string(nvbit_get_func_name(ctx, func, true));
  if (active_from_start && !should_trace_kernel(ctx_kernelid[ctx], func_name))
    active_region = false;
}

void nvbit_at_cuda_event(hipCtx_t ctx, int is_exit, nvbit_api_cuda_t cbid,
                         const char *name, void *params, hipError_t *pStatus) {
  if (skip_flag)
    return;

  if (first_call == true) {
    first_call = false;
    std::string traces_folder = user_folder + "/traces";
    if (mkdir(traces_folder.c_str(), S_IRWXU | S_IRWXG | S_IROTH | S_IXOTH) ==
        -1) {
      if (errno == EEXIST) {
        // alredy exists
      } else {
        // something else
        std::cout << "cannot create folder error:" << strerror(errno)
                  << std::endl;
        return;
      }
    }
    kernelsFile = fopen(ctx_kernelslist[ctx].c_str(), "w");
    statsFile = fopen(ctx_stats_location[ctx].c_str(), "w");
    fprintf(statsFile,
            "kernel id, kernel mangled name, grid_dimX, grid_dimY, grid_dimZ, "
            "#blocks, block_dimX, block_dimY, block_dimZ, #threads, "
            "total_insts, total_reported_insts\n");
    fclose(statsFile);
  }

  switch (cbid) {
  // We start with recording kernel launch events
  // Identify all the possible CUDA launch events without stream
  // parameters, they will not get involved with cuda graph
  case API_CUDA_cuLaunch:
  case API_CUDA_cuLaunchGrid: {
    cuLaunch_params *p = (cuLaunch_params *)params;
    hipFunction_t func = p->f;
    if (!is_exit) {
      enter_kernel_launch(ctx, func, cbid, params, false, false);
    } else {
      leave_kernel_launch(ctx, func);
    }
  } break;
  // To support kernel launched by cuda graph (in addition to existing kernel
  // launche method), we need to do:
  //
  // 1. instrument kernels at hipGraphAddKernelNode event. This is for cases
  // that kernels are manually added to a cuda graph.
  // 2. distinguish captured kernels when kernels are recorded to a graph
  // using stream capture. hipStreamIsCapturing() tells us whether a stream
  // is capturiong.
  // 3. per-kernel instruction counters, since cuda graph can launch multiple
  // kernels at the same time.
  //
  // Three cases:
  //
  // 1. original kernel launch:
  //     1a. for any kernel launch without using a stream, we instrument it
  //     before it is launched, call hipDeviceSynchronize after it is
  //     launched and read the instruction counter of the kernel.
  //     1b. for any kernel launch using a stream, but the stream is not
  //     capturing, we do the same thing as 1a.
  //
  //  2. cuda graph using stream capturing: if a kernel is launched in a
  //  stream and the stream is capturing. We instrument the kernel before it
  //  is launched and do nothing after it is launched, because the kernel is
  //  not running until hipGraphLaunch. Instead, we issue a
  //  hipStreamSynchronize after hipGraphLaunch is done and reset the
  //  instruction counters, since a cloned graph might be launched afterwards.
  //
  //  3. cuda graph manual: we instrument the kernel added by
  //  hipGraphAddKernelNode and do the same thing for hipGraphLaunch as 2.
  //
  // The above method should handle most of cuda graph launch cases.
  // kernel launches with stream parameter, they can be used for cuda graph
  case API_CUDA_cuLaunchKernel_ptsz:
  case API_CUDA_cuLaunchKernel:
  case API_CUDA_cuLaunchCooperativeKernel:
  case API_CUDA_cuLaunchCooperativeKernel_ptsz:
  case API_CUDA_cuLaunchKernelEx:
  case API_CUDA_cuLaunchKernelEx_ptsz:
  case API_CUDA_cuLaunchGridAsync: {
    hipFunction_t func;
    hipStream_t hStream;

    if (cbid == API_CUDA_cuLaunchKernelEx_ptsz ||
        cbid == API_CUDA_cuLaunchKernelEx) {
      cuLaunchKernelEx_params *p = (cuLaunchKernelEx_params *)params;
      func = p->f;
      hStream = p->config->hStream;
    } else if (cbid == API_CUDA_cuLaunchKernel_ptsz ||
               cbid == API_CUDA_cuLaunchKernel ||
               cbid == API_CUDA_cuLaunchCooperativeKernel_ptsz ||
               cbid == API_CUDA_cuLaunchCooperativeKernel) {
      cuLaunchKernel_params *p = (cuLaunchKernel_params *)params;
      func = p->f;
      hStream = p->hStream;
    } else {
      cuLaunchGridAsync_params *p = (cuLaunchGridAsync_params *)params;
      func = p->f;
      hStream = p->hStream;
    }

    hipStreamCaptureStatus streamStatus;
    /* check if the stream is capturing, if yes, do not sync */
    CUDA_SAFECALL(hipStreamIsCapturing(hStream, &streamStatus));
    if (!is_exit) {
      bool stream_capture = (streamStatus == hipStreamCaptureStatusActive);
      enter_kernel_launch(ctx, func, cbid, params, stream_capture);
    } else {
      if (streamStatus != hipStreamCaptureStatusActive) {
        if (verbose >= 1) {
          printf("kernel %s not captured by cuda graph\n",
                 nvbit_get_func_name(ctx, func));
        }
        leave_kernel_launch(ctx, func);
      } else {
        if (verbose >= 1) {
          printf("kernel %s captured by cuda graph\n",
                 nvbit_get_func_name(ctx, func));
        }
      }
    }
  } break;
  case API_CUDA_cuGraphAddKernelNode: {
    cuGraphAddKernelNode_params *p = (cuGraphAddKernelNode_params *)params;
    hipFunction_t func = p->nodeParams->func;

    if (!is_exit) {
      // cuGraphAddKernelNode_params->nodeParams is the same as
      // cuLaunchKernel_params up to sharedMemBytes
      enter_kernel_launch(ctx, func, cbid, (void *)p->nodeParams, false, true);
    }
  } break;
  case API_CUDA_cuGraphLaunch: {
    // if we are exiting a cuda graph launch:
    // Wait until the graph is completed using
    // hipStreamSynchronize()
    if (is_exit) {
      cuGraphLaunch_params *p = (cuGraphLaunch_params *)params;

      CUDA_SAFECALL(hipStreamSynchronize(p->hStream));
      assert(hipGetLastError() == hipSuccess);
      /* push a flush channel kernel */
      skip_flag = true;
      flush_channel<<<1, 1, 0, p->hStream>>>();
      CUDA_SAFECALL(hipStreamSynchronize(p->hStream));
      skip_flag = false;
      assert(hipGetLastError() == hipSuccess);
    }

  } break;
  // Now we need to record cuda memcpy events
  case API_CUDA_cuMemcpyHtoD_v2: {
    if (!is_exit) {
      cuMemcpyHtoD_v2_params *p = (cuMemcpyHtoD_v2_params *)params;
      char buffer[1024];
      kernelsFile = fopen(ctx_kernelslist[ctx].c_str(), "a");
      sprintf(buffer, "MemcpyHtoD,0x%016llx,%llu", p->dstDevice, p->ByteCount);
      fprintf(kernelsFile, buffer);
      fprintf(kernelsFile, "\n");
      fclose(kernelsFile);
    }
  } break;
  // For cuProfiler, we need to set the active region accordingly
  case API_CUDA_cuProfilerStart: {
    if (is_exit && !active_from_start) {
      active_region = true;
    }
  } break;
  case API_CUDA_cuProfilerStop: {
    if (is_exit && !active_from_start) {
      active_region = false;
    }
  } break;
  default:
    break;
  }
}

bool is_number(const std::string &s) {
  std::string::const_iterator it = s.begin();
  while (it != s.end() && std::isdigit(*it))
    ++it;
  return !s.empty() && it == s.end();
}

unsigned get_datawidth_from_opcode(const std::vector<std::string> &opcode) {
  for (unsigned i = 0; i < opcode.size(); ++i) {
    if (is_number(opcode[i])) {
      return (std::stoi(opcode[i], NULL) / 8);
    } else if (opcode[i][0] == 'U' && is_number(opcode[i].substr(1))) {
      // handle the U* case
      unsigned bits;
      sscanf(opcode[i].c_str(), "U%u", &bits);
      return bits / 8;
    }
  }

  return 4; // default is 4 bytes
}

bool check_opcode_contain(const std::vector<std::string> &opcode,
                          std::string param) {
  for (unsigned i = 0; i < opcode.size(); ++i)
    if (opcode[i] == param)
      return true;

  return false;
}

bool base_stride_compress(const uint64_t *addrs, const std::bitset<32> &mask,
                          uint64_t &base_addr, int &stride) {
  // calulcate the difference between addresses
  // write cosnsctive addresses with constant stride in a more
  // compressed way (i.e. start adress and stride)
  bool const_stride = true;
  bool first_bit1_found = false;
  bool last_bit1_found = false;

  for (int s = 0; s < 32; s++) {
    if (mask.test(s) && !first_bit1_found) {
      first_bit1_found = true;
      base_addr = addrs[s];
      if (s < 31 && mask.test(s + 1))
        stride = addrs[s + 1] - addrs[s];
      else {
        const_stride = false;
        break;
      }
    } else if (first_bit1_found && !last_bit1_found) {
      if (mask.test(s)) {
        if (stride != addrs[s] - addrs[s - 1]) {
          const_stride = false;
          break;
        }
      } else
        last_bit1_found = true;
    } else if (last_bit1_found) {
      if (mask.test(s)) {
        const_stride = false;
        break;
      }
    }
  }

  return const_stride;
}

void base_delta_compress(const uint64_t *addrs, const std::bitset<32> &mask,
                         uint64_t &base_addr, std::vector<long long> &deltas) {
  // save the delta from the previous address
  bool first_bit1_found = false;
  uint64_t last_address = 0;
  for (int s = 0; s < 32; s++) {
    if (mask.test(s) && !first_bit1_found) {
      base_addr = addrs[s];
      first_bit1_found = true;
      last_address = addrs[s];
    } else if (mask.test(s) && first_bit1_found) {
      deltas.push_back(addrs[s] - last_address);
      last_address = addrs[s];
    }
  }
}

void *recv_thread_fun(void *args) {
  hipCtx_t ctx = (hipCtx_t)args;
  char *recv_buffer = (char *)malloc(CHANNEL_SIZE);
  while (recv_thread_started) {
    uint32_t num_recv_bytes = 0;
    if (recv_thread_receiving &&
        (num_recv_bytes = channel_host.recv(recv_buffer, CHANNEL_SIZE)) > 0) {
      uint32_t num_processed_bytes = 0;
      while (num_processed_bytes < num_recv_bytes) {
        inst_trace_t *ma = (inst_trace_t *)&recv_buffer[num_processed_bytes];

        /* when we get this cta_id_x it means the kernel has completed
         */
        if (ma->cta_id_x == -1) {
          recv_thread_receiving = false;
          break;
        }

        fprintf(ctx_resultsFile[ctx], "%d ", ma->cta_id_x);
        fprintf(ctx_resultsFile[ctx], "%d ", ma->cta_id_y);
        fprintf(ctx_resultsFile[ctx], "%d ", ma->cta_id_z);
        fprintf(ctx_resultsFile[ctx], "%d ", ma->warpid_tb);
        if (print_core_id) {
          fprintf(ctx_resultsFile[ctx], "%d ", ma->sm_id);
          fprintf(ctx_resultsFile[ctx], "%d ", ma->warpid_sm);
        }
        if (lineinfo) {
          fprintf(ctx_resultsFile[ctx], "%d ", ma->line_num);
        }
        fprintf(ctx_resultsFile[ctx], "%04x ", ma->vpc); // Print the virtual PC
        fprintf(ctx_resultsFile[ctx], "%08x ",
                ma->active_mask & ma->predicate_mask);
        if (ma->GPRDst >= 0) {
          fprintf(ctx_resultsFile[ctx], "1 ");
          fprintf(ctx_resultsFile[ctx], "R%d ", ma->GPRDst);
        } else
          fprintf(ctx_resultsFile[ctx], "0 ");

        // Print the opcode.
        fprintf(ctx_resultsFile[ctx], "%s ",
                id_to_opcode_map[ma->opcode_id].c_str());
        unsigned src_count = 0;
        for (int s = 0; s < MAX_SRC; s++) // GPR srcs count.
          if (ma->GPRSrcs[s] >= 0)
            src_count++;
        fprintf(ctx_resultsFile[ctx], "%d ", src_count);

        for (int s = 0; s < MAX_SRC; s++) // GPR srcs.
          if (ma->GPRSrcs[s] >= 0)
            fprintf(ctx_resultsFile[ctx], "R%d ", ma->GPRSrcs[s]);

        // print addresses
        std::bitset<32> mask(ma->active_mask & ma->predicate_mask);
        if (ma->is_mem) {
          std::istringstream iss(id_to_opcode_map[ma->opcode_id]);
          std::vector<std::string> tokens;
          std::string token;
          while (std::getline(iss, token, '.')) {
            if (!token.empty())
              tokens.push_back(token);
          }
          fprintf(ctx_resultsFile[ctx], "%d ",
                  get_datawidth_from_opcode(tokens));

          bool base_stride_success = false;
          uint64_t base_addr = 0;
          int stride = 0;
          std::vector<long long> deltas;

          if (enable_compress) {
            // try base+stride format
            base_stride_success =
                base_stride_compress(ma->addrs, mask, base_addr, stride);
            if (!base_stride_success) {
              // if base+stride fails, try base+delta format
              base_delta_compress(ma->addrs, mask, base_addr, deltas);
            }
          }

          if (base_stride_success && enable_compress) {
            // base + stride format
            fprintf(ctx_resultsFile[ctx], "%u 0x%llx %d ",
                    address_format::base_stride, base_addr, stride);
          } else if (!base_stride_success && enable_compress) {
            // base + delta format
            fprintf(ctx_resultsFile[ctx], "%u 0x%llx ",
                    address_format::base_delta, base_addr);
            for (int s = 0; s < deltas.size(); s++) {
              fprintf(ctx_resultsFile[ctx], "%lld ", deltas[s]);
            }
          } else {
            // list all the addresses
            fprintf(ctx_resultsFile[ctx], "%u ", address_format::list_all);
            for (int s = 0; s < 32; s++) {
              if (mask.test(s))
                fprintf(ctx_resultsFile[ctx], "0x%016lx ", ma->addrs[s]);
            }
          }
        } else {
          fprintf(ctx_resultsFile[ctx], "0 ");
        }

        // Print the immediate
        fprintf(ctx_resultsFile[ctx], "%d ", ma->imm);

        fprintf(ctx_resultsFile[ctx], "\n");

        num_processed_bytes += sizeof(inst_trace_t);
      }
    }
  }
  free(recv_buffer);
  return NULL;
}

void nvbit_tool_init(hipCtx_t ctx) {
  recv_thread_started = true;
  channel_host.init(0, CHANNEL_SIZE, &channel_dev, NULL);
  pthread_create(&recv_thread, NULL, recv_thread_fun, ctx);
}

void nvbit_at_ctx_term(hipCtx_t ctx) {
  if (recv_thread_started) {
    recv_thread_started = false;
    pthread_join(recv_thread, NULL);
  }
}

void nvbit_at_ctx_init(hipCtx_t ctx) {
  // Everytime we init a context, add the foldername and kernelid to the set
  char buffer[2048];
  sprintf(buffer, "kernelslist_ctx_0x%lx", ctx);
  std::string tmp_kernelslist = user_folder + "/traces/" + buffer;
  ctx_kernelslist[ctx] = tmp_kernelslist;
  sprintf(buffer, "stats_ctx_0x%lx", ctx);
  std::string tmp_stats = user_folder + "/traces/" + buffer;
  ctx_stats_location[ctx] = tmp_stats;
  ctx_kernelid[ctx] = 1;
}